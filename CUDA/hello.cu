
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void)
{
    printf("Hello World from GPU!\n");
    printf("%d", threadIdx.x);
}
int main(void)
{
    printf("Hello World from CPU!\n");

    helloFromGPU <<<1, 10>>>();
    hipDeviceReset();
}